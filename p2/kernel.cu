﻿/**
* ARQUITECTURA DE COMPUTADORES
* 2º Grado en Ingenieria Informatica
*
* Básico 2
*
* Alumno: Rodrigo Pascual Arnaiz 
* Fecha: 26/09/2022
*
*/

///////////////////////////////////////////////////////////////////////////
// includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
 

///////////////////////////////////////////////////////////////////////////
// defines


///////////////////////////////////////////////////////////////////////////
// declaracion de funciones
// HOST: funcion llamada desde el host y ejecutada en el host

/**
* Funcion: propiedadesDispositivo
* Objetivo: Mustra las propiedades del dispositvo, esta funcion
*   es ejecutada llamada y ejecutada desde el host
*
* Param: INT id_dispositivo -> ID del dispotivo
* Return: void
*/
__host__ void propiedadesDispositivo(int id_dispositivo, int &cantidad_computo )
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, id_dispositivo);

    // calculo del numero de cores (SP)
    int cuda_cores = 0;
    int multi_processor_count = deviceProp.multiProcessorCount;
    int major = deviceProp.major;
    cantidad_computo = major;
    int minor = deviceProp.minor;


    switch (major)
    {
    case 1:
        //TESLA
        cuda_cores = 8;
        break;
    case 2:
        //FERMI
        if (minor == 0)
            cuda_cores = 32;
        else
            cuda_cores = 48;
        break;
    case 3:
        //KEPLER
        cuda_cores = 192;
        break;
    case 5:
        //MAXWELL
        cuda_cores = 128;
        break;
    case 6:
        //PASCAL
        cuda_cores = 64;
        break;
    case 7:
        //VOLTA
        cuda_cores = 64;
        break;
    case 8:
        //AMPERE
        cuda_cores = 128;
        break;
    default:
        //DESCONOCIDA
        cuda_cores = 0;
    }

    if (cuda_cores == 0)
    {
        printf("!!!!!dispositivo desconocido!!!!!\n");
    }
    // presentacion de propiedades
    printf("***************************************************\n");
    printf("DISPOSIRIVO %d: %s\n", id_dispositivo, deviceProp.name);
    printf("***************************************************\n");
    printf("> Capacidad de Computo \t\t\t: %d.%d\n", major, minor);
    printf("> N. de MultiProcesadores \t\t: %d \n", multi_processor_count);
    printf("> N. de CUDA Cores (%dx%d) \t\t: %d \n", cuda_cores, multi_processor_count, cuda_cores * multi_processor_count);
    printf("> Memoria Global (total) \t\t: %zu MiB\n", deviceProp.totalGlobalMem / (1 << 20));
    printf("> Memoria Compartida (por bloque) \t: %zu KiB\n", deviceProp.sharedMemPerBlock /
        (1 << 10));
    printf("> Memoria Constante (total) \t\t: %zu KiB\n", deviceProp.totalConstMem / (1 << 10));
    printf("***************************************************\n");
}

/**
* Funcion: rellenarVectorHst
* Objetivo: Funcion que rellena un array pasado por parametro 
*   con numero aleatorios del 0 al 9
*
* Param: INT* arr -> Puntero del array a rellenar
* Param: INT size -> Longitud del array
* Return: void
*/
__host__ void rellenarVectorHst( int *arr, int size ) 
{
    for ( size_t i = 0; i < size; i++ )
    {
        arr[i] = rand() % 10 ;
    }
}

/**
* Funcion: invertirVector
* Objetivo: Funcion que da la vuelta a un vector pasado por paramtro
*
* Param: INT* arr -> Puntero del array a invertir
* Param: INT size -> Longitud del array
* Return: void
*/
__global__ void invertirVector( int *arr, int size)
{
   
    int temporal;
    for ( int i = 0, x = size - 1; i < x; i++, x-- ) {
        temporal = arr[ i ];
        arr[ i ] = arr[ x ];
        arr[x] = temporal;
    }

   
}

/**
* Funcion: sumarArrays
* Objetivo: Funcion que da la vuelta a un vector pasado por paramtro
*
* Param: INT* primer_array -> Primer puntero del array que se quiere sumar  
* Param: INT* segundo_array -> Segundo puntero del array que se quiere sumar  
* Param: INT* array_sumado -> Puntero del array que va a contener el resultado 
* Return: void
*/
__global__ void sumarArrays( int* primer_array, int* segundo_array, int* array_sumado ) 
{
    int id = threadIdx.x;
    array_sumado[id] = primer_array[id] + segundo_array[id];
}


// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{
    // Semilla de random aleatoria 
    srand(time(NULL));

    int cantidad_computo;

    // Obetener el dispisivo cuda
    int numero_dispositivos;
    hipGetDeviceCount(&numero_dispositivos);
    if (numero_dispositivos != 0)
    {
        printf("Se han encontrado <%d> dispositivos CUDA:\n", numero_dispositivos);
        for (int i = 0; i < numero_dispositivos; i++)
        {
            propiedadesDispositivo( i, cantidad_computo );
        }
    }
    else
    {
        printf("!!!!!ERROR!!!!!\n");
        printf("Este ordenador no tiene dispositivo de ejecucion CUDA\n");
        printf("<pulsa [INTRO] para finalizar>");
        getchar();
        return 1;
    }
    
    // declaracion de variables
    int* hst_vector1, * hst_vector2, * hst_resultado;
    int* dev_vector1, * dev_vector2, * dev_resultado;

    int numero_elementos;
    bool is_numero_valido = false;
    bool is_cantidad_valida = false;

    do {

        do {

            printf("Introduce el numero de elementos: ");
            is_numero_valido = scanf( "%i", &numero_elementos );
            printf("\n");

        } while ( !is_numero_valido );

        if ( ( cantidad_computo > 1 && is_numero_valido < 512 ) || ( cantidad_computo > 2 && is_numero_valido < 1024 ) )
        {
            is_cantidad_valida = true;
        }else {
            printf("> ERROR: numero maximo de hilosd superado! [ %d hilos ]\n", cantidad_computo > 1 ? 512 : 1024 );
        }

    } while ( !is_cantidad_valida );

    printf("> Vector de %d elementos \n", is_numero_valido);
    printf("> Lanzamiento con 1 bloque de %d \n", numero_elementos);
    
    

    // reserva de memoria en el host
    hst_vector1 = ( int* )malloc( numero_elementos * sizeof( int ) );
    hst_vector2 = ( int* )malloc( numero_elementos * sizeof( int ) );
    hst_resultado = ( int* )malloc( numero_elementos * sizeof( int ) );

    // reserva de memoria en el device
    hipMalloc( ( void** )&dev_vector1, numero_elementos * sizeof( int ) );
    hipMalloc( ( void** )&dev_vector2, numero_elementos * sizeof( int ) );
    hipMalloc( ( void** )&dev_resultado, numero_elementos * sizeof( int ) );

  
    // Rellenamos el vector con la funcion previamente creada
    rellenarVectorHst( hst_vector1, numero_elementos);

    // Copiamos el vector 1 en el device 2, esto es necesario ya que desde la funcion invertir solo podemos acceder 
    // a la memopria del reservada en el device 
    hipMemcpy( dev_vector2, hst_vector1, numero_elementos * sizeof( int ), hipMemcpyHostToDevice );
    // Invertimos el vector y ese mismo vector es el resultado
    invertirVector<<<1,1>>>( dev_vector2, numero_elementos );
    // Copiamos el contenido del vector device 2 al vector host 2
    hipMemcpy( hst_vector2, dev_vector2, numero_elementos * sizeof( int ), hipMemcpyDeviceToHost );
   

    // Mostrar vector 1
    printf( "VECTOR 1:\n" );
    for ( int i = 0; i < numero_elementos; i++ )
    {
        printf( "%i ", hst_vector1[ i ] );
    }
    printf( "\n" );

    // Mostrar vector 2
    printf("VECTOR 2:\n");
    for ( int i = 0; i < numero_elementos; i++ )
    {
        printf( "%i ", hst_vector2[ i ] );
    }
    printf("\n");


    // Sumar V1 + V2, aqui sucede lo mismo que antes para sumar los dos vectores es necesario 
    // copiar el contenido del vector host 1 a un vector que se encuentre en la memoria del device 
    // en este caso he utilizado la variable  dev_vector1
    hipMemcpy( dev_vector1, hst_vector1, numero_elementos * sizeof( int ), hipMemcpyHostToDevice );
    // Para sumar los dos vectores en vez de usar un bucle for he utilizado los N hilos 
    // siendo N el numero de huecos introducidos por el usuario 
    // para hacer esto es necesario comporbar que no sobrepasamos en numero de hilos 
    sumarArrays<<<1,numero_elementos>>>(dev_vector1, dev_vector2, dev_resultado );
    hipMemcpy( hst_resultado, dev_resultado, numero_elementos * sizeof( int ), hipMemcpyDeviceToHost );

    // Mostrar resultado de la suma
    printf( "\nSUMA:\n", numero_elementos );
    for ( int i = 0; i < numero_elementos; i++ )
    {
        printf( "%i ", hst_resultado[ i ] );
    }
    printf("\n");



    // Salida del programa 
    time_t fecha;
    time(&fecha);
    printf("***************************************************\n");
    printf("Programa ejecutado el: %s\n", ctime(&fecha));
    printf("<pulsa [INTRO] para finalizar>");
    getchar();
    return 0;
}